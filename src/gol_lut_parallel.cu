#include "hip/hip_runtime.h"
/* ECE 5720 Final Project Game of Life Parallel LUT Version
 * Parallel version using Lookup Table
 * 
 * Authors: Eric Tang (et396), Xiaoyu Yan (xy97)
 * Date:    10 May 2020 
 */

#include <stdint.h>
#include "utils.h"
#include "test_case_bits.h"


/* Each byte is 8 cells, each cell is one bit
 * curr_world  : shared array for the entire grid
 * next_world  : next shared array for the entire grid
 * num_bytes   : number of bytes to iterate through for this kernel,
 *               each byte is 8 cells
 * world_length: length of the ????  
 */
__global__ void gol_lut_cycle( uint8_t *curr_world, uint8_t *next_world, uint64_t num_bytes, 
                           uint64_t world_length, uint64_t arr_length ){
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  // int offset = blockDim.x * gridDim.x;
  register uint8_t curr_8_states, curr_bit = 0, next_8_states = 0,
                   num_alive = 0, north_byte, south_byte, threshold = 4;
  uint8_t NE_byte, SE_byte, NW_byte, SW_byte;
  uint64_t start = x*num_bytes; 
  uint64_t end   = start + num_bytes;   
  // printf("nb[%d] s[%d] e[%d] x:[%d] \n", num_bytes, start, end, x);                       
  for ( uint64_t i = start; i < end; i++ ){
    curr_8_states = curr_world[i];
    next_8_states = curr_8_states;
    for (int bit = 0; bit < 8; bit ++){
      // iterate through each bit
      curr_bit = (curr_8_states >> bit) & 0x1; // extract cell state 
      num_alive = 0;
      // Check west and east first since they are in register
      // look west in same register
      if ( bit != 7 )
        if ( (curr_8_states >> (bit+1)) & 0x1 ) num_alive++;
      
      // look east in same register
      if ( bit != 0 )
        if ( (curr_8_states >> (bit-1)) & 0x1 ) num_alive++;
        
      // look north
      if ( i > (world_length - 1) ){
        north_byte = curr_world[i-world_length];
        if ( (north_byte >> (bit))   & 0x1 ) num_alive++;
        if ( bit == 0 ){
          if ( (north_byte >> (bit+1)) & 0x1 ) num_alive++;
        }
        else if ( bit == 7 ){
          if ( (north_byte >> (bit-1)) & 0x1 ) num_alive++;
        }
        else{
          if ( (north_byte >> (bit+1)) & 0x1 ) num_alive++;
          if ( (north_byte >> (bit-1)) & 0x1 ) num_alive++;
        }
      } 

      // look south but don't have to if already past threshold
      if ( (i < (arr_length - world_length)) && (num_alive < threshold) ){
        south_byte = curr_world[i+world_length];
        if ( (south_byte >> (bit))   & 0x1 ) num_alive++;
        if ( bit == 0 ){
          if ( (south_byte >> (bit+1)) & 0x1 ) num_alive++;
        }
        else if ( bit == 7 ){
          if ( (south_byte >> (bit-1)) & 0x1 ) num_alive++;
        }
        else{
          if ( (south_byte >> (bit-1)) & 0x1 ) num_alive++;
          if ( (south_byte >> (bit+1)) & 0x1 ) num_alive++;
        }
      }

      // look west in diff reg
      if ( (bit == 7) && (i % world_length != 0) && (num_alive < threshold) ){
        // for the lsb
        if ( curr_world[i-1] & 0x1 ) num_alive++;
        NW_byte = curr_world[i-world_length-1];
        if ( NW_byte & 0x1 ) num_alive++;
        SW_byte = curr_world[i+world_length-1];
        if ( SW_byte & 0x1 ) num_alive++;
      }
      
      // look east in diff reg
      if ( (bit == 0) && ((i + 1) % world_length != 0) && (num_alive < threshold) ){
        // for the msb
        if ( (curr_world[i-1] >> 7) & 0x1 ) num_alive++;
        NE_byte = curr_world[i-world_length+1];
        if ( (NE_byte >> 7) & 0x1 ) num_alive++;
        SE_byte = curr_world[i+world_length+1];
        if ( (SE_byte >> 7) & 0x1 ) num_alive++;
      }

      // Determine if an alive cell will die 
      if (curr_bit && (num_alive < 2 || num_alive > 3)) 
        next_8_states ^= (uint8_t)(1 << bit);

      // Determine if a dead cell will be alive
      else if ( !curr_bit && (num_alive == 3))
        next_8_states ^= (uint8_t)(1 << bit);
    }
    next_world[i] = next_8_states;
  }
}

/* Run Game of Life Simulation 
 * return 1 if error occurs
 */
int gol_lut( uint8_t *world, uint64_t N, uint64_t P, int rounds, int test, 
             uint8_t **ref, int trace ){

	/* Set grid parameters */
	uint64_t world_length = N/8;
  	uint64_t num_elements = N*N/8;
  	struct timespec t_start, t_end;
	long double average = 0, st, ed, diff;
  	int blocks = P / 1024;

  	dim3 Block(blocks); // Square pattern
  	dim3 Grid(P);

  	uint8_t *dev_curr_world, *dev_next_world;
  	hipMalloc((void **) &dev_curr_world, num_elements*sizeof(uint8_t)); 
  	hipMalloc((void **) &dev_next_world, num_elements*sizeof(uint8_t)); 
  
	for ( int i = 0; i < rounds; i++ ){
    	clock_gettime(CLOCK_MONOTONIC, &t_start); /* Start Timer */
    	hipMemcpy(dev_curr_world, world, num_elements*sizeof(uint8_t), hipMemcpyHostToDevice);
    	gol_lut_cycle<<<Grid, Block>>>(dev_curr_world, dev_next_world, N*N/8/P, 
                                       world_length, num_elements);
    	hipMemcpy(world, dev_next_world, num_elements*sizeof(uint8_t), hipMemcpyDeviceToHost);
    	clock_gettime(CLOCK_MONOTONIC, &t_end);   /* End timer */
    	st = t_start.tv_sec + (long double)t_start.tv_nsec/BILLION;
    	ed = t_end.tv_sec + (long double)t_end.tv_nsec/BILLION;
    	diff = ed - st;
    	average += diff/((long double)rounds);
    	if (test && !world_bits_correct(world, ref[i], N)) 
			return 1;
    	if (trace) 
			print_world_bits(world, N);
  	}
  	printf("Grid Size: %ldx%ld, # Rounds: %d, # Threads: %ld\n", N, N, rounds, P*blocks);
  	printf("Average time per round: %.13LFs\n", average);
  	return 0;
}


int main( int argc, char** argv ){

	// Default values
  	int test   = 0; // Run direct test cases
  	uint64_t N = 8; // Matrix size 
  	uint64_t P = 1; // number of threads
 	int ROUNDS = 5; // Number of Rounds
  	int trace  = 0; // print trace of world  

	/* Set Game of Life parameters acording to user input */
  	if ( argc > 1 ) test = atoi(argv[1]); 
  	if ( argc > 2 ) {
    	N = atoi(argv[2]); // Dimensions of the block
    	if ( N % 8 != 0 ){
      		printf( "Invalid N:[%ld]; must be divisible by 8\n", N );
      		N = 8;
    	}	 
  	}
	if ( argc > 3 ) { 
    	P = atoi(argv[3]); // number of threads
    	if ( P > N*N/8 ){
      		printf( "Invalid P:[%ld]; Too many threads for number of elements %ld\n", P, N*N/8 );
      		return 1;
    	}
    	if ( N*N/8 % P != 0 ){
      		printf( "Invalid P:[%ld]; Number of threads should be a factor of %ld\n", P, N*N/8 );
      		return 1;
    	}
  	}
  	if ( argc > 4 ) ROUNDS = atoi(argv[4]); 
  	if ( argc > 5 ) trace  = atoi(argv[5]); 

  	uint8_t *world, **ref;
	
  	if (test){
		/* Setup and run all test cases*/
    	int num_correct = 0;
    	N      = T_DIM;
    	ROUNDS = T_ROUNDS - 1;

    	// Test 1
    	printf("Running test 1\n");
    	world  = test_1[0];
    	ref    = (uint8_t**) malloc( sizeof(uint8_t*) * ROUNDS );
    	for ( int r = 0; r < ROUNDS; r++ )
      		ref[r] = test_1[r+1];
    	if (!gol_lut( world, N, P, ROUNDS, test, ref, trace )) 
			num_correct++;

    	// Test 2
    	printf("Running test 2\n");
    	world  = test_2[0];
    	for ( int r = 0; r < ROUNDS; r++ )
      		ref[r] = test_2[r+1];
    	if (!gol_lut( world, N, P, ROUNDS, test, ref, trace )) 
			num_correct++;

    	// Test 3
    	printf("Running test 3\n");
    	world  = test_3[0];
    	for ( int r = 0; r < ROUNDS; r++ )
      		ref[r] = test_3[r+1];
    	if (!gol_lut( world, N, P, ROUNDS, test, ref, trace )) 
			num_correct++;

    	// Test 4
    	printf("Running test 4\n");
    	world  = test_4[0];
    	for ( int r = 0; r < ROUNDS; r++ )
      		ref[r] = test_4[r+1];
    	if (!gol_lut( world, N, P, ROUNDS, test, ref, trace )) 
			num_correct++;

    	// Test 5
    	printf("Running test 5\n");
    	world  = test_5[0];
    	for ( int r = 0; r < ROUNDS; r++ )
      		ref[r] = test_5[r+1];
    	if (!gol_lut( world, N, P, ROUNDS, test, ref, trace )) 
			num_correct++;

		// Test 6
    	printf("Running test 6\n");
    	world  = test_6[0];
    	for ( int r = 0; r < ROUNDS; r++ )
      		ref[r] = test_6[r+1];
    	if (!gol_lut( world, N, P, ROUNDS, test, ref, trace )) 
			num_correct++;

    	// Test 7
    	printf("Running test 7\n");
    	world  = test_7[0];
    	for ( int r = 0; r < ROUNDS; r++ )
      		ref[r] = test_7[r+1];
    	if (!gol_lut( world, N, P, ROUNDS, test, ref, trace )) 
			num_correct++;

    	// Test 8
    	printf("Running test 8\n");
    	world  = test_8[0];
    	for ( int r = 0; r < ROUNDS; r++ )
      		ref[r] = test_8[r+1];
    	if (!gol_lut( world, N, P, ROUNDS, test, ref, trace )) 
			num_correct++;

    	// Test 9
    	printf("Running test 9\n");
    world  = test_9[0];
    N      = T9_DIM;
    ROUNDS = T9_ROUNDS - 1;
    for ( int r = 0; r < ROUNDS; r++ )
      ref[r] = test_9[r+1];
    if (!gol_lut( world, N, P, ROUNDS, test, ref, trace )) num_correct++;
    
    printf("%s %d/9 tests passed %s\n", KBLU, num_correct, 9, KNRM);
    free(ref);

  } else {
	/* Run simulation w/ random seed */
    srand48(1);
    uint64_t n_elements = N*N/8;
    world = (uint8_t*)malloc(n_elements * sizeof(uint8_t));
    for(int r = 0; r < N; r++){
      for(int c = 0; c < N/8; c++)
        world[r*N/8+c] = (uint8_t)(rand() % 256);
    } 
    ref   = NULL;
    if (trace) print_world_bits( world, N );
    gol_lut( world, N, P, ROUNDS, test, ref, trace );
    free(world);
  }
  return 0;
}

